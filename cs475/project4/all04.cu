
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <ctime>
#include <sys/time.h>
#include <sys/resource.h>
#include <omp.h>

// SSE stands for Streaming SIMD Extensions

#define SSE_WIDTH	4
#define ALIGNED		__attribute__((aligned(16)))


#define NUMTRIES	200

#ifndef DATASET_SIZE
#define DATASET_SIZE	1024*1024
#endif

#ifndef THREADS_PER_BLOCK
#define THREADS_PER_BLOCK 128
#endif

ALIGNED float A[DATASET_SIZE];
ALIGNED float B[DATASET_SIZE];
ALIGNED float C[DATASET_SIZE];


void	SimdMul(    float *, float *,  float *, int );
__global__ void	NonSimdMul( float *, float *,  float *, int );
float	SimdMulSum(    float *, float *, int );
__global__ void	NonSimdMulSum( float *, float *, float *, int );

void    CudaCheckError( );

int
main( int argc, char *argv[ ] )
{
	for( int i = 0; i < DATASET_SIZE; i++ )
	{
		A[i] = sqrtf( (float)(i+1) );
		B[i] = sqrtf( (float)(i+1) );
	}
    float *dA, *dB, *dC;
    hipMalloc( (void **)(&dA), sizeof(A) );
    hipMalloc( (void **)(&dB), sizeof(B) );
    hipMalloc( (void **)(&dC), sizeof(C) );
    CudaCheckError( );

    // copy host memory to the device:
    hipMemcpy( dA, A, DATASET_SIZE*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dB, B, DATASET_SIZE*sizeof(float), hipMemcpyHostToDevice );
    CudaCheckError( );

    // setup the execution parameters:
    dim3 grid( DATASET_SIZE / THREADS_PER_BLOCK, 1, 1 );
    dim3 threads( THREADS_PER_BLOCK, 1, 1 );

	fprintf( stderr, "%12d\t", DATASET_SIZE );

	double maxPerformance = 0.;
	for( int t = 0; t < NUMTRIES; t++ )
	{
		double time0 = omp_get_wtime( );
        // create and start the timer:
        hipDeviceSynchronize( );
        // allocate the events that we'll use for timing:
        hipEvent_t start, stop;
        hipEventCreate( &start );
        hipEventCreate( &stop );
        CudaCheckError( );
        // record the start event:
        hipEventRecord( start, NULL );
        CudaCheckError( );

		NonSimdMul<<<grid, threads>>>( dA, dB, dC, DATASET_SIZE );


		double time1 = omp_get_wtime( );
		double perf = (double)DATASET_SIZE / (time1 - time0);
		if( perf > maxPerformance )
			maxPerformance = perf;
	}
	double megaMults = maxPerformance / 1000000.;
	fprintf( stderr, "N %10.2lf\t", megaMults );
	double mmn = megaMults;


	maxPerformance = 0.;
	for( int t = 0; t < NUMTRIES; t++ )
	{
		double time0 = omp_get_wtime( );
		SimdMul( A, B, C, DATASET_SIZE );
		double time1 = omp_get_wtime( );
		double perf = (double)DATASET_SIZE / (time1 - time0);
		if( perf > maxPerformance )
			maxPerformance = perf;
	}
	megaMults = maxPerformance / 1000000.;
	fprintf( stderr, "S %10.2lf\t", megaMults );
	double mms = megaMults;
	double speedup = mms/mmn;
	fprintf( stderr, "(%6.2lf)\t", speedup );


	maxPerformance = 0.;
	float sumn, sums;
	for( int t = 0; t < NUMTRIES; t++ )
	{
		double time0 = omp_get_wtime( );
        // create and start the timer:
        hipDeviceSynchronize( );
        // allocate the events that we'll use for timing:
        hipEvent_t start, stop;
        hipEventCreate( &start );
        hipEventCreate( &stop );
        CudaCheckError( );
        // record the start event:
        hipEventRecord( start, NULL );
        CudaCheckError( );

		NonSimdMulSum<<<grid, threads>>>( dA, dB, dC, DATASET_SIZE );

        hipMemcpy( dA, A, DATASET_SIZE*sizeof(float), hipMemcpyDeviceToHost);
        CudaCheckError();
		double time1 = omp_get_wtime( );
		double perf = (double)DATASET_SIZE / (time1 - time0);
		if( perf > maxPerformance )
			maxPerformance = perf;
	}
	double megaMultAdds = maxPerformance / 1000000.;
	fprintf( stderr, "N %10.2lf\t", megaMultAdds );
	mmn = megaMultAdds;


	maxPerformance = 0.;
	for( int t = 0; t < NUMTRIES; t++ )
	{
		double time0 = omp_get_wtime( );
		sums = SimdMulSum( A, B, DATASET_SIZE );
		double time1 = omp_get_wtime( );
		double perf = (double)DATASET_SIZE / (time1 - time0);
		if( perf > maxPerformance )
			maxPerformance = perf;
	}
	megaMultAdds = maxPerformance / 1000000.;
	fprintf( stderr, "S %10.2lf\t", megaMultAdds );
	mms = megaMultAdds;
	speedup = mms/mmn;
	fprintf( stderr, "(%6.2lf)\n", speedup );
	//fprintf( stderr, "[ %8.1f , %8.1f , %8.1f ]\n", C[DATASET_SIZE-1], sumn, sums );

	return 0;
}


__global__
void
NonSimdMul( float *dA, float *dB, float *dC, int n )
{
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    if( gid < n )
        dC[gid] = dA[gid] * dB[gid];
}

__global__
void
NonSimdMulSum( float *dA, float *dB, float *dC, int n )
{
    int gid = blockIdx.x*blockDim.x + threadIdx.x;
    if( gid < n )
        dC[gid] = dA[gid] * dB[gid];

    float sum = 0;
    for( int i = 0; i < n; ++i ) sum += dC[i];
    return;
}


__host__
void
SimdMul( float *a, float *b,   float *c,   int len )
{
	int limit = ( len/SSE_WIDTH ) * SSE_WIDTH;
	__asm
	(
		".att_syntax\n\t"
		"movq    -24(%rbp), %r8\n\t"		// a
		"movq    -32(%rbp), %rcx\n\t"		// b
		"movq    -40(%rbp), %rdx\n\t"		// c
	);

	for( int i = 0; i < limit; i += SSE_WIDTH )
	{
		__asm
		(
			".att_syntax\n\t"
			"movups	(%r8), %xmm0\n\t"	// load the first sse register
			"movups	(%rcx), %xmm1\n\t"	// load the second sse register
			"mulps	%xmm1, %xmm0\n\t"	// do the multiply
			"movups	%xmm0, (%rdx)\n\t"	// store the result
			"addq $16, %r8\n\t"
			"addq $16, %rcx\n\t"
			"addq $16, %rdx\n\t"
		);
	}

	for( int i = limit; i < len; i++ )
	{
		c[i] = a[i] * b[i];
	}
}



__host__
float
SimdMulSum( float *a, float *b, int len )
{
	float sum[4] = { 0., 0., 0., 0. };
	int limit = ( len/SSE_WIDTH ) * SSE_WIDTH;

	__asm
	(
		".att_syntax\n\t"
		"movq    -40(%rbp), %r8\n\t"		// a
		"movq    -48(%rbp), %rcx\n\t"		// b
		"leaq    -32(%rbp), %rdx\n\t"		// &sum[0]
		"movups	 (%rdx), %xmm2\n\t"		// 4 copies of 0. in xmm2
	);

	for( int i = 0; i < limit; i += SSE_WIDTH )
	{
		__asm
		(
			".att_syntax\n\t"
			"movups	(%r8), %xmm0\n\t"	// load the first sse register
			"movups	(%rcx), %xmm1\n\t"	// load the second sse register
			"mulps	%xmm1, %xmm0\n\t"	// do the multiply
			"addps	%xmm0, %xmm2\n\t"	// do the add
			"addq $16, %r8\n\t"
			"addq $16, %rcx\n\t"
		);
	}

	__asm
	(
		".att_syntax\n\t"
		"movups	 %xmm2, (%rdx)\n\t"	// copy the sums back to sum[ ]
	);

	for( int i = limit; i < len; i++ )
	{
		sum[0] += a[i] * b[i];
	}

	return sum[0] + sum[1] + sum[2] + sum[3];
}

void
CudaCheckError( ) {
    return;
    hipError_t e = hipGetLastError( );
    if( e != hipSuccess ) {
        fprintf( stderr, "CUDA failure %s:%d: '%s'\n", __FILE__,
                __LINE__, hipGetErrorString(e));
    }
}

